#include "hip/hip_runtime.h"
﻿#include "SF_CUDA.cuh"
#include "Math_Helper.cuh"

// Host variables
Person* cells;

// Device variables
Person* deviceCells;
int* debugDevice;
int* debugHost;

// 1 block = 1 cell
dim3 blocksPerGrid(CELLS_PER_AXIS, CELLS_PER_AXIS);

// 32 Threads per block/cell, 3x3 for main cell + neighbors
dim3 threadsPerBlock(MAX_OCCUPATION, 3, 3);

// function to add the elements of two arrays
__global__ void debug(Person* grid_ptr, int* debugVal)
{
	for (int i = 0; i < SPAWNED_ACTORS; i++)
	{
		grid_ptr[i].state = 2;
	}

	//atomicAdd(debugVal, 1);
	atomicMax(debugVal, blockIdx.x * 10 + blockIdx.y);

	//grid_ptr[0].velocity = calculateSF(&grid_ptr[0], &grid_ptr[1]);
}

__device__ float2 calculateSF(Person* personA, Person* personB)
{
	float v_a0 = magnitude(personA->velocity);
	float v_b0 = magnitude(personB->velocity);

	if (v_a0 * v_a0 < 0.001f || v_b0 * v_b0 < 0.001f)
	{
		return make_float2(0.f, 0.f);
	}

	float2 dir_a = personA->goal - personA->position;
	float2 e_a = dir_a / magnitude(dir_a);

	float2 dir_b = personB->goal - personB->position;
	float2 e_b = dir_b / magnitude(dir_b);

	float2 e2 = EPSILON * v_a0 * e_a - v_b0 * e_b;
	e2 = normalize(e2);
	float2 e1 = make_float2(e2.y, -e2.x);

	const float2 r_ab = personA->position - personB->position;
	float e1_result = dot(r_ab, e1);
	e1_result *= e1_result;
	float e2_result = dot(r_ab, e2);
	e2_result *= e2_result;

	float gamma_a = dot(r_ab, e2) >= 0.f ? THETA : 1 + DELTA * v_a0;

	float V_ab = S * std::powf(EULER, -std::sqrtf(e1_result + e2_result / (gamma_a * gamma_a)) / R);

	float2 f_ab = make_float2(-r_ab.x * V_ab, -r_ab.y * V_ab);

	return f_ab;
}

__global__ void calculateCellForce(Person* device_grid, int* debugVal)
{
	__shared__ float2 totalForces[MAX_OCCUPATION][9];

	short2 cellAPos = make_short2(blockIdx.x, blockIdx.y);
	short cellA = cellPosToCell(cellAPos);

	Person* personA = &device_grid[cellA * MAX_OCCUPATION + threadIdx.x];
	if(personA->state == FREE)
		return;

	short2 cellBPos = make_short2(cellAPos.x - 1 + threadIdx.y, cellAPos.y - 1 + threadIdx.z);

	short cellB = cellPosToCell(cellBPos);
	float2 forceVector = make_float2(0.f, 0.f);

	if (!(cellB < 0 || cellB >= CELLS_PER_AXIS * CELLS_PER_AXIS))
	{
		// People in analyzed cell
		int blockppl = 0;
		// Iterate over space in neighbor cell
		for (int i = 0; i < MAX_OCCUPATION; i++)
		{
			// Ignore yourself
			if (threadIdx.y == 1 && threadIdx.z == 1 && threadIdx.x % MAX_OCCUPATION == i)
				continue;

			Person* other = &device_grid[cellB * MAX_OCCUPATION + i];

			if (other->state == FREE)
				continue;

			forceVector = forceVector + calculateSF(personA, other);
			blockppl++;
		}

		// People in main/influenced cell
		int ppl = maskToInt(__ballot_sync(0xFFFFFFFF, personA->state == OCCUPIED));

		if ((threadIdx.y != 1 || threadIdx.z != 1) && (blockppl > 20 || ppl > 26))
		{
			forceVector.x -= (threadIdx.y - 1) * (blockppl - 20) * AVOIDANCE_FORCE;
			forceVector.y -= (threadIdx.z - 1) * (blockppl - 20) * AVOIDANCE_FORCE;
		}
	}

	totalForces[threadIdx.x][threadIdx.y + threadIdx.z * 3] = forceVector;

	__syncthreads();

	if (threadIdx.y == 1 && threadIdx.z == 1)
	{
		float2 resultForce = make_float2(0.f, 0.f);
		for (int i = 0; i < 9; i++)
		{
			if (isnan(totalForces[threadIdx.x][i]))
				continue;

			resultForce = resultForce + totalForces[threadIdx.x][i];
		}

		personA->velocity = make_float2(personA->velocity.x - resultForce.x, personA->velocity.y - resultForce.y);

		float2 newPos = personA->position + personA->velocity * DELTA;

		// Check if person moves to other cell
		int oldCell = posToCell(personA->position.x, personA->position.y);
		int newCell = posToCell(newPos.x, newPos.y);

		if (oldCell != newCell)
		{
			bool cellChanged = false;
			//bool moveSuccessful = reserveSpace(device_grid, newCell, cellA * MAX_OCCUPATION + threadIdx.x);

			// Look for space in new cell
			for (int i = newCell * MAX_OCCUPATION; i < (newCell + 1) * MAX_OCCUPATION; i++)
			{
				if (atomicCAS(&device_grid[i].state, FREE, RESERVED) == FREE)
				{
					device_grid[cellA * MAX_OCCUPATION + threadIdx.x].state = LEAVING;

					device_grid[i] = Person(device_grid[cellA * MAX_OCCUPATION + threadIdx.x]);
					device_grid[i].state = RESERVED;

					cellChanged = true;
					break;
				}
			}

			if (!cellChanged)
			{
				personA->velocity = make_float2(0.f, 0.f);
			}
		}
	}
}

__global__ void completeMove(Person* device_grid, int* debugVal)
{
	short2 cellAPos = make_short2(blockIdx.x, blockIdx.y);
	short cellA = cellPosToCell(cellAPos);

	for (int i = 0; i < MAX_OCCUPATION; i++)
	{
		Person* personA = &device_grid[cellA * MAX_OCCUPATION + i];

		if (personA->state == FREE)
			continue;

		if (personA->state == LEAVING)
		{
			personA->state = FREE;
			personA->velocity = make_float2(0.f, 0.f);
			continue;
		}

		if (personA->state == RESERVED)
		{
			personA->state = OCCUPIED;
		}

		atomicAdd(debugVal, 1);
		
		personA->position = personA->position + personA->velocity * DELTA;

		float2 goalDir = make_float2(
			personA->goal.x - personA->position.x,
			personA->goal.y - personA->position.y);

		if (magnitude(goalDir) < MIN_DIST)
		{
			personA->goal = personA->position; //make_float2(0.f, 0.f); //getRandomPos();
			personA->velocity = make_float2(0.f, 0.f);
			personA->direction = make_float2(0.f, 0.f);
		}
		else
		{
			goalDir = normalize(goalDir);
			personA->direction = goalDir;
			personA->velocity = goalDir * SPEED;
		}
	}
}

__device__ bool reserveSpace(int newCell, int oldIndex)
{
	bool cellChanged = false;


	return cellChanged;
}

__device__ int posToCell(int x, int y)
{
	int cellX = x / CELL_SIZE;
	int cellY = y / CELL_SIZE;
	return cellX + cellY * CELLS_PER_AXIS;
}

int toIndexH(int x, int y)
{
	int cellX = x / CELL_SIZE;
	int cellY = y / CELL_SIZE;
	return cellX + cellY * CELLS_PER_AXIS;
}

bool addToGrid(Person p)
{
	int cell = toIndexH(p.position.x, p.position.y);

	if(cell >= TOTAL_CELLS)
	{
		return false;
	}
	
	bool placed = false;

	for (int i = 0; i < MAX_OCCUPATION; i++)
	{
		int index = cell * MAX_OCCUPATION + i;

		if (cells[index].state != FREE)
			continue;

		cells[index] = Person(p);
		placed = true;
		break;
	}

	//if (placed)
	//	std::cout << "added to cell " << cell << "\n";

	return placed;
}

void init()
{
	cells = static_cast<Person*>(malloc(sizeof(Person) * TOTAL_SPACES));
	for (int i = 0; i < TOTAL_SPACES; i++)
	{
		cells[i] = Person();
	}

	int totallySpawned = 0;
	int remainingSpawns = SPAWNED_ACTORS;
	float spacing = .7f;
	for(int x = 0; x < CELLS_PER_AXIS; x++)
	{
		int posX = x * CELL_SIZE;
		for(int y = 0; y < CELLS_PER_AXIS; y++)
		{
			int posY = y * CELL_SIZE;
			for(int i = 0; i < 1; i++)
			{
				//float2 spawnPos = make_float2(posX + i * spacing, posY + i * spacing);
				//addToGrid(Person(spawnPos, getRandomPos()));

				float2 spawnPos = make_float2(posX + i / 4, posY + i % 4);
				addToGrid(Person(spawnPos, getRandomPos()));

				totallySpawned++;
				if(--remainingSpawns <= 0)
					goto endspawn;
			}
		}
	}

	endspawn:
	std::cout << "Spawned " << totallySpawned << " people.\n";

	hipMalloc((void**)&deviceCells, TOTAL_SPACES * sizeof(Person));
	hipMemcpy(deviceCells, cells, TOTAL_SPACES * sizeof(Person), hipMemcpyHostToDevice);

	int temp = 0;
	debugHost = &temp;

	debugDevice = nullptr;
	hipMalloc((void**)&debugDevice, sizeof(int));
	hipMemcpy(debugDevice, debugHost, sizeof(int), hipMemcpyHostToDevice);
}

void initTest()
{
	cells = static_cast<Person*>(malloc(sizeof(Person) * TOTAL_SPACES));
	for (int i = 0; i < TOTAL_SPACES; i++)
	{
		cells[i] = Person();
	}

	addToGrid(Person(make_float2(1, 1), make_float2(20, 1)));
	addToGrid(Person(make_float2(20, 1.5), make_float2(1, 1.5)));
	//addToGrid(Person(make_float2(1, 7), make_float2(18, 7)));
	addToGrid(Person(make_float2(1, 13), make_float2(18, 13)));

	addToGrid(Person(make_float2(5.f, 15.f), make_float2(20.f, 20.f)));
	addToGrid(Person(make_float2(22.f, 22.f), make_float2(5.f, 15.f)));

	hipMalloc((void**)&deviceCells, TOTAL_SPACES * sizeof(Person));
	hipMemcpy(deviceCells, cells, TOTAL_SPACES * sizeof(Person), hipMemcpyHostToDevice);

	int temp = 0;
	debugHost = &temp;

	debugDevice = nullptr;
	hipMalloc((void**)&debugDevice, sizeof(int));
	hipMemcpy(debugDevice, debugHost, sizeof(int), hipMemcpyHostToDevice);
}

void close()
{
	// Free memory
	//free(cells);
	//delete debugHost;

	hipFree(deviceCells);
	hipFree(debugDevice);
}

int simulate()
{
	*debugHost = 0;
	hipMemcpy(debugDevice, debugHost, sizeof(int), hipMemcpyHostToDevice);
	calculateCellForce << < blocksPerGrid, threadsPerBlock >> >(deviceCells, debugDevice);
	hipDeviceSynchronize();

	completeMove << < blocksPerGrid, 1 >> >(deviceCells, debugDevice);
	hipDeviceSynchronize();

	hipMemcpy(cells, deviceCells, TOTAL_SPACES * sizeof(Person), hipMemcpyDeviceToHost);
	hipMemcpy(debugHost, debugDevice, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "Simulated " << *debugHost << " people.\n";
	/*
	int people[5] = {0,0,0,0,0};
	for(int i = 0; i < TOTAL_SPACES; i++)
	{
		switch(cells[i].state)
		{
			case FREE: people[0]++;
			break;
			case OCCUPIED: people[1]++;
			break;
			case RESERVED: people[2]++;
			break;
			case LEAVING: people[3]++;
			break;
			default: people[4]++;
		}
	}

	std::cout << "Total: " << people[0] << " FREE | " << people[1] << " OCCUPIED | " <<
		people[2] << " RESERVED | " << people[3] << " LEAVING | " << people[4] << " OTHER!\n";
	
	*/
	return 0;
}

std::vector<PersonVisuals> convertToVisual(bool debugPrint)
{
	std::vector<PersonVisuals> persons;
	int addedActors = 0;

	for (int i = 0; i < TOTAL_SPACES; i++)
	{
		Person& p = cells[i];
		if (p.state != FREE)
		{
			float2 dir = p.direction;
			dir.y = -dir.y;
			persons.push_back(PersonVisuals(simToGL(p.position), dir));

			if (++addedActors >= DRAWN_ACTORS)
			{
				break;
			}
		}
	}

	return persons;
}

float2 simToGL(float2 pos)
{
	float maxVal = CELLS_PER_AXIS * CELL_SIZE;
	float xPos = pos.x / maxVal * 2.f - 1.f;
	float yPos = (pos.y / maxVal * 2.f - 1.f) * -1.f;

	return make_float2(xPos, yPos);
}

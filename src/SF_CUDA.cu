#include "hip/hip_runtime.h"
﻿#include "SF_CUDA.cuh"
#include <chrono>
#include <iostream>
#include "Math_Helper.cuh"

namespace SF_CUDA
{
	// Host variables
	Person* cells;

	// Device variables
	Person* deviceCells;

	// 1 (thread) block = 1 cell
	dim3 blocksPerGrid(CELLS_PER_AXIS, CELLS_PER_AXIS, 1);
	
	// Per space: 9 threads, 1 for each influencing cell
	dim3 threadsPerBlock(MAX_OCCUPATION, 3, 3);

	__device__ float2 calculateSF(Person* personA, Person* personB)
	{
		float v_a0 = magnitude(personA->velocity);
		float v_b0 = magnitude(personB->velocity);

		if (v_a0 * v_a0 < 0.001f || v_b0 * v_b0 < 0.001f)
		{
			return make_float2(0.f, 0.f);
		}

		float2 dir_a = personA->goal - personA->position;
		float2 e_a = dir_a / magnitude(dir_a);

		float2 dir_b = personB->goal - personB->position;
		float2 e_b = dir_b / magnitude(dir_b);

		float2 e2 = EPSILON * v_a0 * e_a - v_b0 * e_b;
		e2 = normalize(e2);
		float2 e1 = make_float2(e2.y, -e2.x);

		const float2 r_ab = personA->position - personB->position;
		float e1_result = dot(r_ab, e1);
		e1_result *= e1_result;
		float e2_result = dot(r_ab, e2);
		e2_result *= e2_result;

		float gamma_a = dot(r_ab, e2) >= 0.f ? THETA : 1 + DELTA * v_a0;

		float V_ab = S * std::powf(EULER, -std::sqrtf(e1_result + e2_result / (gamma_a * gamma_a)) / R);

		float2 f_ab = make_float2(-r_ab.x * V_ab, -r_ab.y * V_ab);

		return f_ab;
	}

	__global__ void calculateCellForce(Person* device_grid)
	{
		// Saves forces of each cell on influenced person
		__shared__ float2 totalForces[MAX_OCCUPATION][9];

		short2 cellAPos = make_short2(blockIdx.x, blockIdx.y);
		int cellA = cellPosToIndex(cellAPos);

		// Influenced person/space
		Person* personA = &device_grid[cellA * MAX_OCCUPATION + threadIdx.x];

		// If space is empty, thread can terminate early
		if (personA->state != OCCUPIED)
			return;

		short2 cellBPos = make_short2(cellAPos.x - 1 + threadIdx.y, cellAPos.y - 1 + threadIdx.z);

		int cellB = cellPosToIndex(cellBPos);
		float2 forceVector = make_float2(0.f, 0.f);

		if (cellB >= 0 && cellB < CELLS_PER_AXIS * CELLS_PER_AXIS)
		{
			// Number of people in influencing cell, important for congestion avoidance
			int blockppl = 0;

			// Iterate over space in neighbor cell
			for (int i = 0; i < MAX_OCCUPATION; i++)
			{
				// Ignore yourself
				if (threadIdx.y == 1 && threadIdx.z == 1 && threadIdx.x % MAX_OCCUPATION == i)
					continue;

				Person* other = &device_grid[cellB * MAX_OCCUPATION + i];

				if (other->state != OCCUPIED)
					continue;

				forceVector = forceVector + calculateSF(personA, other);
				blockppl++;
			}

			// Number of people in influenced cell
			int ppl = mask_to_int(__ballot_sync(0xFFFFFFFF, personA->state == OCCUPIED));

			// Only calculate avoidance force if influencing cell =/= influenced cell
			if ((threadIdx.y != 1 || threadIdx.z != 1) && (blockppl > 20 || ppl > 26))
			{
				forceVector.x -= (threadIdx.y - 1.f) * (blockppl - 20) * AVOIDANCE_FORCE;
				forceVector.y -= (threadIdx.z - 1.f) * (blockppl - 20) * AVOIDANCE_FORCE;
			}
		}

		// Save calculated force in shared memory
		totalForces[threadIdx.x][threadIdx.y + threadIdx.z * 3] = forceVector;

		// Wait for all threads to complete calculation
		__syncthreads();

		// Only center cell sums and applies all social forces
		if (threadIdx.y == 1 && threadIdx.z == 1)
		{
			float2 resultForce = make_float2(0.f, 0.f);
			for (int i = 0; i < 9; i++)
			{
				if (float2_isnan(totalForces[threadIdx.x][i]))
					continue;

				resultForce = resultForce + totalForces[threadIdx.x][i];
			}

			personA->updateVelocity(personA->velocity - resultForce * DELTA);

			float2 newPos = personA->position + personA->velocity * DELTA;

			// Check if person moves to other cell
			int oldCell = personPosToCellIndex(personA->position.x, personA->position.y);
			int newCell = personPosToCellIndex(newPos.x, newPos.y);

			if (oldCell != newCell)
			{
				bool reservedSpace = false;

				if (newCell >= 0 && newCell < CELLS_PER_AXIS * CELLS_PER_AXIS)
				{
					// Look for space in new cell
					for (int i = newCell * MAX_OCCUPATION; i < (newCell + 1) * MAX_OCCUPATION; i++)
					{
						if (atomicCAS(&device_grid[i].state, FREE, RESERVED) == FREE)
						{
							device_grid[cellA * MAX_OCCUPATION + threadIdx.x].state = LEAVING;

							device_grid[i] = Person(device_grid[cellA * MAX_OCCUPATION + threadIdx.x]);
							device_grid[i].state = RESERVED;

							reservedSpace = true;

							break;
						}
					}
				}

				// If entry to other cell was denied, block movement
				if (!reservedSpace)
				{
					personA->velocity = make_float2(0.f, 0.f);
				}
			}
		}
	}

	__global__ void completeMove(Person* device_grid)
	{
		int cell = cellPosToIndex(blockIdx.x, blockIdx.y);
		Person* person = &device_grid[cell * MAX_OCCUPATION + threadIdx.x];

		// Terminate early if space is empty
		if (person->state == FREE)
			return;

		// Mark space as FREE again and terminate
		if (person->state == LEAVING)
		{
			person->state = FREE;
			return;
		}

		// If person moved to other cell, mark space as OCCUPIED
		if (person->state == RESERVED)
		{
			person->state = OCCUPIED;
		}

		// Update position
		person->position = person->position + person->velocity * DELTA;

		// Update direction to goal
		float2 goalDir = make_float2(
			person->goal.x - person->position.x,
			person->goal.y - person->position.y);

		goalDir = normalize(goalDir);
		person->direction = goalDir;

		person->updateVelocity(goalDir * SPEED);
	}

	bool add_to_grid(const Person& p)
	{
		int cell = cellPosToIndex(p.position / CELL_SIZE);

		for (int i = 0; i < MAX_OCCUPATION; i++)
		{
			int index = cell * MAX_OCCUPATION + i;

			if (cells[index].state != FREE)
				continue;

			cells[index] = Person(p);
			return true;
		}

		return false;
	}

	void init()
	{
		cells = static_cast<Person*>(malloc(sizeof(Person) * CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION));
		for (int i = 0; i < CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION; i++)
		{
			cells[i] = Person();
		}

		int totallySpawned = 0;

		for (int i = 0; i < SPAWNED_ACTORS; i++)
		{
			bool spawned = false;
			while (!spawned)
			{
				spawned = add_to_grid(Person(getRandomPos(), getRandomPos()));
			}

			totallySpawned++;
		}

		std::cout << "Spawned " << totallySpawned << " people.\n";

		hipError_t error = hipMalloc((void**)&deviceCells, CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION * sizeof(Person));
		if (error)
			std::cout << "Error while allocating CUDA memory\n";

		std::cout << "Allocated " << CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION * sizeof(Person) / 1024 / 1024 << " MB on GPU\n";

		hipMemcpy(deviceCells, cells, CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION * sizeof(Person), hipMemcpyHostToDevice);
	}

	double timeH2D = 0.f;
	double timeD2H = 0.f;
	int transfersMeasured = 0;

	void simulate()
	{
		auto t1 = std::chrono::high_resolution_clock::now();
		hipMemcpy(deviceCells, cells, CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION * sizeof(Person), hipMemcpyHostToDevice);
		auto t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> ms_double = t2 - t1;
		timeH2D += ms_double.count();

		calculateCellForce << < blocksPerGrid, threadsPerBlock >> > (deviceCells);

		hipError_t error = hipDeviceSynchronize();
		if (error)
		{
			std::cout << "CalculateForce: " << hipGetErrorName << ": " << hipGetErrorString(error) << "\n";
		}

		completeMove << < blocksPerGrid, MAX_OCCUPATION >> > (deviceCells);
		hipDeviceSynchronize();

		error = hipDeviceSynchronize();
		if (error)
		{
			std::cout << "CompleteMove: " << hipGetErrorName << ": " << hipGetErrorString(error) << "\n";
		}

		t1 = std::chrono::high_resolution_clock::now();
		hipMemcpy(cells, deviceCells, CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION * sizeof(Person), hipMemcpyDeviceToHost);
		t2 = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> ms_double2 = t2 - t1;
		timeD2H += ms_double2.count();

		transfersMeasured++;
	}

	void printTransferTime()
	{
		std::cout << "Avg. Host to Device = " << timeH2D / transfersMeasured << "\n";
		std::cout << "Avg. Device to Host = " << timeD2H / transfersMeasured << "\n";
	}

	std::vector<PersonVisuals> convertToVisual()
	{
		std::vector<PersonVisuals> persons;

		for (int i = 0; i < CELLS_PER_AXIS * CELLS_PER_AXIS * MAX_OCCUPATION; i++)
		{
			Person& p = cells[i];
			if (p.state != FREE)
			{
				if (dist(p.position, p.goal) < MIN_DIST)
				{
					p.goal = getRandomPos();
				}

				float2 dir = p.direction;
				dir.y = -dir.y;

				persons.emplace_back(simCoordToGL(p.position), dir);
			}
		}
		
		return persons;
	}
}
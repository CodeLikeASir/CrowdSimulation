#include "hip/hip_runtime.h"
﻿#include "SF_CUDA.cuh"

#include <iostream>

#include "Math_Helper.cuh"

namespace SF_CUDA
{
	// Host variables
	Person* cells;

	// Device variables
	Person* deviceCells;

	// 1 (thread) block = 1 cell
	dim3 blocksPerGrid(CELLS_PER_AXIS, CELLS_PER_AXIS, 1);

	// Per cell: 1 thread per space,
	// Per space: 9 threads, 1 for each influencing cell
	dim3 threadsPerBlock(MAX_OCCUPATION, 3, 3);
	
	__device__ float2 calculateSF(Person* personA, Person* personB)
	{
		float v_a0 = magnitude(personA->velocity);
		float v_b0 = magnitude(personB->velocity);

		if (v_a0 * v_a0 < 0.001f || v_b0 * v_b0 < 0.001f)
		{
			return make_float2(0.f, 0.f);
		}

		float2 dir_a = personA->goal - personA->position;
		float2 e_a = dir_a / magnitude(dir_a);

		float2 dir_b = personB->goal - personB->position;
		float2 e_b = dir_b / magnitude(dir_b);

		float2 e2 = EPSILON * v_a0 * e_a - v_b0 * e_b;
		e2 = normalize(e2);
		float2 e1 = make_float2(e2.y, -e2.x);

		const float2 r_ab = personA->position - personB->position;
		float e1_result = dot(r_ab, e1);
		e1_result *= e1_result;
		float e2_result = dot(r_ab, e2);
		e2_result *= e2_result;

		float gamma_a = dot(r_ab, e2) >= 0.f ? THETA : 1 + DELTA * v_a0;

		float V_ab = S * std::powf(EULER, -std::sqrtf(e1_result + e2_result / (gamma_a * gamma_a)) / R);

		float2 f_ab = make_float2(-r_ab.x * V_ab, -r_ab.y * V_ab);

		return f_ab;
	}

	__global__ void calculateCellForce(Person* device_grid)
	{
		// Saves forces of each cell on influenced person
		__shared__ float2 totalForces[MAX_OCCUPATION][9];

		short2 cellAPos = make_short2(blockIdx.x, blockIdx.y);
		int cellA = cellPosToIndex(cellAPos);

		// Influenced person/space
		Person* personA = &device_grid[cellA * MAX_OCCUPATION + threadIdx.x];

		// If space is empty, thread can terminate early
		if (personA->state == FREE)
			return;

		short2 cellBPos = make_short2(cellAPos.x - 1 + threadIdx.y, cellAPos.y - 1 + threadIdx.z);

		int cellB = cellPosToIndex(cellBPos);
		float2 forceVector = make_float2(0.f, 0.f);

		if (!(cellB < 0 || cellB >= CELLS_PER_AXIS * CELLS_PER_AXIS))
		{
			// Number of people in influencing cell, important for congestion avoidance
			int blockppl = 0;
			
			// Iterate over space in neighbor cell
			for (int i = 0; i < MAX_OCCUPATION; i++)
			{
				// Ignore yourself
				if (threadIdx.y == 1 && threadIdx.z == 1 && threadIdx.x % MAX_OCCUPATION == i)
					continue;

				Person* other = &device_grid[cellB * MAX_OCCUPATION + i];

				if (other->state == FREE)
					continue;

				forceVector = forceVector + calculateSF(personA, other);
				blockppl++;
			}

			// Number of people in influenced cell
			int ppl = mask_to_int(__ballot_sync(0xFFFFFFFF, personA->state == OCCUPIED));

			// Only calculate avoidance force if influencing cell =/= influenced cell
			if ((threadIdx.y != 1 || threadIdx.z != 1) && (blockppl > 20 || ppl > 26))
			{
				forceVector.x -= (threadIdx.y - 1) * (blockppl - 20) * AVOIDANCE_FORCE;
				forceVector.y -= (threadIdx.z - 1) * (blockppl - 20) * AVOIDANCE_FORCE;
			}
		}

		// Save calculated force in shared memory
		totalForces[threadIdx.x][threadIdx.y + threadIdx.z * 3] = forceVector;

		// Wait for all threads to complete calculation
		__syncthreads();

		// Only center cell sums and applies all social forces
		if (threadIdx.y == 1 && threadIdx.z == 1)
		{
			float2 resultForce = make_float2(0.f, 0.f);
			for (int i = 0; i < 9; i++)
			{
				if (float2_isnan(totalForces[threadIdx.x][i]))
					continue;

				resultForce = resultForce + totalForces[threadIdx.x][i];
			}

			personA->velocity = make_float2(personA->velocity.x - resultForce.x, personA->velocity.y - resultForce.y);

			float2 newPos = personA->position + personA->velocity * DELTA;

			// Check if person moves to other cell
			int oldCell = personPosToCellIndex(personA->position.x, personA->position.y);
			int newCell = personPosToCellIndex(newPos.x, newPos.y);

			if (oldCell != newCell)
			{
				bool cellChanged = false;

				// Look for space in new cell
				for (int i = newCell * MAX_OCCUPATION; i < (newCell + 1) * MAX_OCCUPATION; i++)
				{
					if (atomicCAS(&device_grid[i].state, FREE, RESERVED) == FREE)
					{
						device_grid[cellA * MAX_OCCUPATION + threadIdx.x].state = LEAVING;

						device_grid[i] = Person(device_grid[cellA * MAX_OCCUPATION + threadIdx.x]);
						device_grid[i].state = RESERVED;

						cellChanged = true;
						break;
					}
				}

				// If entry to other cell was denied, block movement
				if (!cellChanged)
				{
					personA->velocity = make_float2(0.f, 0.f);
				}
			}
		}
	}

	__global__ void completeMove(Person* device_grid)
	{
		int cell = cellPosToIndex(blockIdx.x, blockIdx.y);
		Person* person = &device_grid[cell * MAX_OCCUPATION + threadIdx.x];

		// Terminate early if space is empty
		if (person->state == FREE)
			return;

		// Mark space as FREE again and terminate
		if (person->state == LEAVING)
		{
			person->state = FREE;
			return;
		}

		// If person moved to other cell, mark space as OCCUPIED
		if (person->state == RESERVED)
		{
			person->state = OCCUPIED;
		}

		// Update position
		person->position = person->position + person->velocity * DELTA;

		// Update direction to goal
		float2 goalDir = make_float2(
			person->goal.x - person->position.x,
			person->goal.y - person->position.y);

		// If close enough to goal, stop moving
		if (magnitude(goalDir) < MIN_DIST)
		{
			person->goal = person->position;
			person->velocity = make_float2(0.f, 0.f);
			person->direction = make_float2(0.f, 0.f);
		}
		// Otherwise update move direction and velocity
		else
		{
			goalDir = normalize(goalDir);
			person->direction = goalDir;
			person->velocity = goalDir * SPEED;
		}
	}

	void add_to_grid(const Person& p)
	{
		int cell = cellPosToIndex(p.position / CELL_SIZE);
		
		//cell_coords.x + cell_coords.y * CELLS_PER_AXIS;

		for (int i = 0; i < MAX_OCCUPATION; i++)
		{
			int index = cell * MAX_OCCUPATION + i;

			if (cells[index].state != FREE)
				continue;

			cells[index] = Person(p);
			break;
		}
	}

	void init()
	{
		cells = static_cast<Person*>(malloc(sizeof(Person) * TOTAL_SPACES));
		for (int i = 0; i < TOTAL_SPACES; i++)
		{
			cells[i] = Person();
		}

		int totallySpawned = 0;
		int remainingSpawns = SPAWNED_ACTORS;

		int spawnsPerRow = ceil(sqrtf(SPAWNED_ACTORS));
		float spacing = CELLS_PER_AXIS * CELL_SIZE / spawnsPerRow;
		for (int x = 0; x < spawnsPerRow; x++)
		{
			for (int y = 0; y < spawnsPerRow; y++)
			{
				float2 spawnPos = make_float2(x * spacing, y * spacing);
				add_to_grid(Person(spawnPos, getRandomPos()));

				totallySpawned++;
				if (--remainingSpawns <= 0)
					goto endspawn;
			}
		}

		endspawn:
		std::cout << "Spawned " << totallySpawned << " people.\n";
		
		hipMalloc((void**)&deviceCells, TOTAL_SPACES * sizeof(Person));
		hipMemcpy(deviceCells, cells, TOTAL_SPACES * sizeof(Person), hipMemcpyHostToDevice);
	}

	void simulate()
	{
		calculateCellForce << < blocksPerGrid, threadsPerBlock >> > (deviceCells);
		
		hipError_t error = hipDeviceSynchronize();
		if (error)
		{
			std::cout << hipGetErrorName << ": " << hipGetErrorString(error) << "\n";
		}

		completeMove << < blocksPerGrid, MAX_OCCUPATION >> > (deviceCells);
		hipDeviceSynchronize();

		hipMemcpy(cells, deviceCells, TOTAL_SPACES * sizeof(Person), hipMemcpyDeviceToHost);
	}

	std::vector<PersonVisuals> convertToVisual()
	{
		std::vector<PersonVisuals> persons;
		int remainingDraws = DRAWN_ACTORS > 0 ? DRAWN_ACTORS : SPAWNED_ACTORS;

		for (int i = 0; i < TOTAL_SPACES; i++)
		{
			Person& p = cells[i];
			if (p.state != FREE)
			{
				float2 dir = p.direction;
				dir.y = -dir.y;
				persons.push_back(PersonVisuals(simCoordToGL(p.position), dir));

				if (--remainingDraws <= 0) break;
			}
		}

		return persons;
	}
}